#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <stdio.h>
#include "opencv2/opencv.hpp"
using namespace cv;
using namespace std;


#define BLUR_SIZE 32

__global__ void rgb2grayincuda(uchar3 *const d_in, unsigned char *const d_out,
							   uint imgheight, uint imgwidth)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < imgwidth && idy < imgheight)
	{
		uchar3 rgb = d_in[idy * imgwidth + idx];
		d_out[idy * imgwidth + idx] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
	}
}


__global__ void blurKernel(uchar3 *const d_in, uchar3 *const d_out,
						   uint imgheight, uint imgwidth)
{
	const unsigned int Col = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int Row = blockIdx.y * blockDim.y + threadIdx.y;

	if (Col < imgheight && Row < imgwidth)
	{
		int pixValx = 0;
		int pixValy = 0;
		int pixValz = 0;
		int pixels = 0;

	
		for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow)
		{
			for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol)
			{
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;


				if (curRow > -1 && curRow < imgheight && curCol > -1 && curCol < imgwidth)
				{
					pixValx += d_in[curRow * imgwidth + curCol].x;
					pixValy += d_in[curRow * imgwidth + curCol].y;
					pixValz += d_in[curRow * imgwidth + curCol].z;
					pixels++; 
				}
			}
		}

		d_out[Row * imgwidth + Col].x = (unsigned char)(pixValx / pixels);
		d_out[Row * imgwidth + Col].y = (unsigned char)(pixValy / pixels);
		d_out[Row * imgwidth + Col].z = (unsigned char)(pixValz / pixels);
	}
}

int main(void)
{
	Mat srcImage = imread("./Lenna.png");
	const uint imgheight = srcImage.rows;
	const uint imgwidth = srcImage.cols;

	Mat grayImage(imgheight, imgwidth, CV_8UC1, Scalar(0));

	uchar3 *d_in;
	unsigned char *d_out;

	hipMalloc((void **)&d_in, imgheight * imgwidth * sizeof(uchar3));
	hipMalloc((void **)&d_out, imgheight * imgwidth * sizeof(unsigned char));

	hipMemcpy(d_in, srcImage.data, imgheight * imgwidth * sizeof(uchar3), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(32, 32);
	dim3 blocksPerGrid((imgwidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
					   (imgheight + threadsPerBlock.y - 1) / threadsPerBlock.y);

	rgb2grayincuda<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, imgheight, imgwidth);

	hipMemcpy(grayImage.data, d_out, imgheight * imgwidth * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_out);

	imwrite("greyImage.jpg", grayImage);


	// blur
	Mat blurImage(imgheight, imgwidth, CV_8UC3);
	uchar3 *d_out2;
	hipMalloc((void **)&d_out2, imgheight * imgwidth * sizeof(uchar3));
	blurKernel<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out2, imgheight, imgwidth);
	hipMemcpy(blurImage.data, d_out2, imgheight * imgwidth * sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out2);
	imwrite("blurImage32.jpg", blurImage);

	return 0;
}